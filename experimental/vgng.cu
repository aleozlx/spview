#include <iostream>
#include <random>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

template<typename T>
struct SOMNetwork {
    size_t units, dims;
    T *weights = nullptr; // W
    bool *connections = nullptr; // C
    unsigned *conn_ages = nullptr; // T
    T *errors = nullptr; // E
};

template<typename T>
void AllocCPU(SOMNetwork<T> &network, size_t units, size_t dims) {
    network.units = units;
    network.dims = dims;
    network.weights = new T[units * dims];
    network.connections = new bool[units * units];
    network.conn_ages = new unsigned[units * units];
    network.errors = new T[units];
}

template<typename T>
void AllocGPU(SOMNetwork<T> &network, size_t units, size_t dims) {
	network.units = units;
	network.dims = dims;
	hipMallocManaged(&network.weights, units*dims * sizeof(T));
	hipMallocManaged(&network.connections,units * units*sizeof(bool));
	hipMallocManaged(&network.conn_ages,units * units*sizeof(unsigned));
	hipMallocManaged(&network.errors,units*sizeof(T));
}

template<typename T>
void DeallocCPU(SOMNetwork<T> &network) {
    delete[] network.weights;
    delete[] network.connections;
    delete[] network.conn_ages;
    delete[] network.errors;
}

template<typename T>
void DeallocGPU(SOMNetwork<T> &network) {
	hipFree(network.weights);
	hipFree(network.connections);
	hipFree(network.conn_ages);
	hipFree(network.errors);
}

template<typename T>
void InitUnified(SOMNetwork<T> &network, const size_t units, const size_t dims) {
	// Network parameters are on unified memory
	std::random_device dev;
	std::mt19937 rng(dev());
	std::uniform_real_distribution<T> dist(-2, 2);
	for (int i = 0; i < units * dims; ++i)
		network.weights[i] = dist(rng);
	for (int i = 0; i < units; ++i)
		for (int j = 0; j < units; ++j)
			network.connections[i * units + j] = i != j;
	for (int i = 0; i < units * units; ++i)
		network.conn_ages = 0;
	for (int i = 0; i < units; ++i)
		network.errors = 0;
}

template<typename T>
void InitCPU(SOMNetwork<T> &network, const size_t units, const size_t dims) {
    AllocCPU(network, units, dims);
	InitUnified(network, units, dims);
}

template<typename T>
void InitGPU(SOMNetwork<T> &network, const size_t units, const size_t dims) {
	AllocGPU(network, units, dims);
	InitUnified(network, units, dims);
}

enum Device { CPU = 1, GPU = 2 };

template<typename T>
class SOMBase {
protected:
    const T *data;
    const size_t samples, dims, units;
    SOMNetwork<T> network;
	const Device device;
public:
    SOMBase(const T *input_data, size_t samples, size_t dims, size_t units, Device dev=GPU) :
            data(input_data), samples(samples), dims(dims), units(units), device(dev) {
		switch (device) {
		case CPU: InitCPU(network, units, dims); break;
		case GPU: InitGPU(network, units, dims); break;
		}
    }

    virtual ~SOMBase() {
		switch (device) {
		case CPU: DeallocCPU(network); break;
		case GPU: DeallocGPU(network); break;
		}
    }

    void UpdateClusters(unsigned outlier_tolerance = 1) {

    }
};

template<typename T>
class GrowingNeuralGas : public SOMBase<T> {
public:
    GrowingNeuralGas(const T *input_data, size_t samples, size_t dims, size_t units, Device dev = GPU):
        SOMBase(input_data, samples, dims, units, dev) {

    }

    virtual void Fit(T e_epsilon, T e_lambda, const unsigned MAX_AGE, const unsigned STEP_NEW_UNIT, T ERR_DECAY_GLOBAL, const unsigned N_PASS) {
        const size_t DIM_DATA = network.dims;
        unsigned sequence = 0;
        for(unsigned p = 0; p<N_PASS; ++p) {
            std::cout<<"    Pass #"<<(p+1)<<std::endl;
            // TODO shuffle data
//            steps =
        }
    }
};

int main(int, char **) {
//    v_network_ctor(5, 5);
    GrowingNeuralGas<float> gng(nullptr, 20, 5, 16);

    return 0;
}
